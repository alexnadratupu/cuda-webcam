#include "hip/hip_runtime.h"
#include "sepia.h"

/**
	\file sepia.cu
	CUDA sepia kernelinin launcher metodunu ve kernelini tan�mlar.
*/

/** Kernel 1 griddeki blok boyutu ( BLOCK_SIZE x BLOCK_SIZE kare bloklar ). */
#define BLOCK_SIZE (32)

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 0

/**	
	G�r�nt�n�n sepia tonlamas�n� hesaplayan kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.
	*/
__global__
void gpuSepia(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	// normalize edilmi� pikselleri 1'den ��kartt���m�zda g�r�nt�n�n negatifini alm�� oluruz.

	float b = *( image + cIdx     );
	float g = *( image + cIdx + 1 );
	float r = *( image + cIdx + 2 );


		/*outputRed = (inputRed * .393) + (inputGreen *.769) + (inputBlue * .189)

outputGreen = (inputRed * .349) + (inputGreen *.686) + (inputBlue * .168)

outputBlue = (inputRed * .272) + (inputGreen *.534) + (inputBlue * .131)*/

	*( image + cIdx     ) = .272f * r + .534f * g + .131f * b; // Red kanal�
	*( image + cIdx + 1 ) = .349f * r + .686f * g + .168f * b; // Green kanal�
	*( image + cIdx + 2 ) = .393f * r + .769f * g + .189f * b; // Blue kanal�
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuSepia kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceSepiaLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuSepia<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuSepia kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}
