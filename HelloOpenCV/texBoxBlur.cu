#include "hip/hip_runtime.h"
#ifndef TEX_INVERT_CU
#define TEX_INVERT_CU

#include "texBoxBlur.h"

texture<float4, 2, hipReadModeElementType> texBlur1;

#define BLOCK_SIZE_X (32)
#define BLOCK_SIZE_Y (32)

#define ENABLE_TIMING_CODE 0

__global__
void gpuTexBoxBlur(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	float tw = 1.0f / width; // Texture kordinatlar�nda 1 pixel geni�lik.
	float th = 1.0f / height; // Texture kordinatlar�nda 1 pixel y�kseklik.

	float tu = ( float )col * tw;
	float tv = ( float )row * th;


	float4 texVal = make_float4(0, 0, 0, 0);

	tw *= 8;
	th *= 8;

#pragma unroll
	for(int i = 0; i < 5; i++)
	{
#pragma unroll
		for(int j = 0; j < 5; j++)
		{			
			texVal += tex2D( texBlur1, tu + ( i - 2 ) * tw, tv + ( j - 2 ) * th );
		}
	}

	texVal *= 1.0f / 25;


	*( image + cIdx )     = texVal.x;
	*( image + cIdx + 1 ) = texVal.y;
	*( image + cIdx + 2 ) = texVal.z;
}

void deviceTexBoxBlurLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE_X, BLOCK_SIZE_Y );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuTexBoxBlur<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuTexBoxBlur kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}


#endif