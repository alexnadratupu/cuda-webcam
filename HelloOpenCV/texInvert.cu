#include "hip/hip_runtime.h"
#include "texInvert.h"

#include "TextureHeader.cu"

#define BLOCK_SIZE (32)

#define ENABLE_TIMING_CODE 0

__global__
void gpuTexInvert(
	float* image,
	int width,
	int height
	)
{
	int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int k = blockIdx.y * BLOCK_SIZE + threadIdx.y;

	int cIdx = (i*width + k) * 3;

	float4 texVal = tex2D(tex, i, k);

	*( image + cIdx ) = 1 - texVal.x;
	*( image + cIdx + 1 ) = 1 - texVal.y;
	*( image + cIdx + 2 ) = 1 - texVal.z;
}

void deviceTexInvertLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( height / dimBlock.x, width / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuTexInvert<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuInvert kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}
