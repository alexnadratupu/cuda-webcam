#include "hip/hip_runtime.h"
#ifndef TEX_INVERT_CU
#define TEX_INVERT_CU

#include "texInvert.h"

//#include "TextureHeader.cu"

texture<float4, 2, hipReadModeElementType> tex;

#define BLOCK_SIZE (32)

#define ENABLE_TIMING_CODE 0

__global__
void gpuTexInvert(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.


	float tu = (float)col / width;
	float tv = (float)row / height;

	//float4 texVal = tex2D( tex, k + .5f, i + .5f );
	float4 texVal = tex2D( tex, tu, tv );

	*( image + cIdx )     = 1 - texVal.x;
	*( image + cIdx + 1 ) = 1 - texVal.y;
	*( image + cIdx + 2 ) = 1 - texVal.z;
}

void deviceTexInvertLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuTexInvert<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuInvert kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}


#endif