#include "hip/hip_runtime.h"
#ifndef TEX_INVERT_CU
#define TEX_INVERT_CU

/**
	\file texInvert.cu
	CUDA texture invert kernelinin launcher metodunu ve kernelini tan�mlar.
*/

#include "texInvert.h"

texture<float4, 2, hipReadModeElementType> texInvert1; /**< Kernelde kullan�lan texture sembol�. */

#define BLOCK_SIZE (32) /**< Blok boyutu ( BLOCK_SIZE x BLOCK_SIZE kare blok ). */

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 0

/**	
	Texture kullanarak g�r�nt�n�n negatifini alan kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.

	*/
__global__
void gpuTexInvert(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.


	float tu = (float)col / width;
	float tv = (float)row / height;

	//float4 texVal = tex2D( tex, k + .5f, i + .5f );
	float4 texVal = tex2D( texInvert1, tu, tv );

	*( image + cIdx )     = 1 - texVal.x;
	*( image + cIdx + 1 ) = 1 - texVal.y;
	*( image + cIdx + 2 ) = 1 - texVal.z;
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuTexInvert kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceTexInvertLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuTexInvert<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuInvert kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}


#endif