#include "hip/hip_runtime.h"
#pragma once

#include "SingleCudaTexFilter.h"

void SingleCudaTexFilter::InitFilter(int width, int height)
{
	SingleCudaFilter::InitFilter(width, height);

	/*
	allocate device texture memory
	*/

	// get texture reference.
	hipGetTextureReference(&constTexRefPtr, HIP_SYMBOL(textureSymbolName));
	checkCUDAError("get texture reference");
	
	texRefPtr = const_cast<textureReference*>( constTexRefPtr );

	channelDesc = hipCreateChannelDesc<float4>();
	hipMallocArray( &cu_array, &texRefPtr->channelDesc, width, height ); 
	checkCUDAError("malloc device image");
 
	/*
	allocate device memory for result.
	*/

	hipMalloc( (void**) &d_Image, 3 * sizeof(float) * width * height );
	checkCUDAError("malloc device image2");

	/*
	allocate host memory
	*/

	hipHostMalloc( (void**) &h_Image, 4 * sizeof(float) * width * height );
	checkCUDAError("malloc host image");


	// bind texture

	// set texture parameters
	texRefPtr->addressMode[0] = hipAddressModeWrap;
	texRefPtr->addressMode[1] = hipAddressModeWrap;

	texRefPtr->filterMode = hipFilterModeLinear;
	texRefPtr->normalized = true;    // access with normalized texture coordinates



	checkCUDAError("FilterImage: Bind Texture");
}

void SingleCudaTexFilter::FilterImage(char* imageData)
{
	int index;
	// copy imageData to GPU.
	for(int i=0; i<4*width*height; i+=4)
	{
		index = (i/4) * 3; // 4.kanal bo�, float4 i�in, kernellerde kullan�lmaz.
		*(h_Image + i) = (unsigned char)*(imageData + index) / 255.0f; // normalize and copy image
		*(h_Image + i + 1) = (unsigned char)*(imageData + index + 1) / 255.0f; // normalize and copy image
		*(h_Image + i + 2) = (unsigned char)*(imageData + index + 2) / 255.0f; // normalize and copy image
		*(h_Image + i + 3) = (unsigned char)0; // normalize and copy image
	}

	
	
	hipMemcpyToArray( cu_array, 0, 0, h_Image, sizeof(float4) * width * height, hipMemcpyHostToDevice);
	checkCUDAError("FilterImage: memcpy");

		// Bind the array to the texture
	hipBindTextureToArray( texRefPtr, cu_array, &texRefPtr->channelDesc );



	// Execute kernel.
	kernelLauncher( d_Image, width, height );
	
	// copy results back to h_C.
	hipMemcpy( h_Image, d_Image, 3 * sizeof(float) * width * height, hipMemcpyDeviceToHost);
	checkCUDAError("FilterImage: memcpy2");

	for(int i=0; i<3*width*height; i++)
	{
		// d_Image, 3 kanall� oldu�u i�in do�rudan imageData'ya h_Image �zerinden kopyalan�r.
		*(imageData + i) = satchar(*(h_Image + i) * 255);
	}
}

void SingleCudaTexFilter::ReleaseFilter()
{
	SingleCudaFilter::ReleaseFilter();

	hipUnbindTexture( texRefPtr );
	checkCUDAError("unbind tex");

	hipFreeArray( cu_array );
	checkCUDAError("free device tex array");
}