#include "hip/hip_runtime.h"
 //hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
 //   hipArray* cu_array;
 //   cutilSafeCall( hipMallocArray( &cu_array, &channelDesc, width, height )); 
 //   cutilSafeCall( hipMemcpyToArray( cu_array, 0, 0, h_data, size, hipMemcpyHostToDevice));

 //   // set texture parameters
 //   tex.addressMode[0] = hipAddressModeWrap;
 //   tex.addressMode[1] = hipAddressModeWrap;
 //   tex.filterMode = hipFilterModeLinear;
 //   tex.normalized = true;    // access with normalized texture coordinates

 //   // Bind the array to the texture
 //   cutilSafeCall( hipBindTextureToArray( tex, cu_array, channelDesc));

#pragma once


#include "SingleCudaProcessor.h"

#include "cudaCommon.h"

#include "TextureHeader.cu"


class SingleCudaTexProcessor : public SingleCudaProcessor
{
private:
	float* h_Image;
	float* d_Image;

	hipArray* cu_array;
	hipChannelFormatDesc channelDesc;

	// Kernel launcher metodu, device pointer, width ve height'a ihtiya� duyar.
	void (*kernelLauncher)(float*, int, int);

public:

	SingleCudaTexProcessor( void kernelLauncher(float*, int, int) )
		: SingleCudaProcessor(kernelLauncher)
	{
	}

	virtual void InitProcessing(int width, int height)
	{
		SingleCudaTexProcessor::InitProcessing(width, height);

		/*
		allocate device texture memory
		*/

		channelDesc = hipCreateChannelDesc<float4>();
		hipMallocArray( &cu_array, &channelDesc, width, height ); 
		checkCUDAError("malloc device image");
 
		/*
		allocate device memory for result.
		*/

		hipMalloc( (void**) &d_Image, 3 * sizeof(float) * width * height );
		checkCUDAError("malloc device image2");

		/*
		allocate host memory
		*/

		hipHostMalloc( (void**) &h_Image, 4 * sizeof(float) * width * height );
		checkCUDAError("malloc host image");
	}

	virtual void ProcessImage(char* imageData)
	{
		int index;
		// copy imageData to GPU.
		for(int i=0; i<4*width*height; i+=4)
		{
			index = (i/4) * 3; // 4.kanal bo�, float4 i�in, kernellerde kullan�lmaz.
			*(h_Image + i) = (unsigned char)*(imageData + index) / 255.0f; // normalize and copy image
			*(h_Image + i + 1) = (unsigned char)*(imageData + index + 1) / 255.0f; // normalize and copy image
			*(h_Image + i + 2) = (unsigned char)*(imageData + index + 2) / 255.0f; // normalize and copy image
			*(h_Image + i + 3) = (unsigned char)0; // normalize and copy image
		}


		hipMemcpyToArray( cu_array, 0, 0, h_Image, 4 * sizeof(float) * width * height, hipMemcpyHostToDevice);
		checkCUDAError("ProcessImage: memcpy");


		// set texture parameters
		tex.addressMode[0] = hipAddressModeWrap;
		tex.addressMode[1] = hipAddressModeWrap;
		tex.filterMode = hipFilterModeLinear;
		tex.normalized = false;    // access with normalized texture coordinates

		// Bind the array to the texture
		hipBindTextureToArray( &tex, cu_array, &channelDesc);
		checkCUDAError("ProcessImage: Bind Texture");
				
		// Execute kernel.
		kernelLauncher( d_Image, width, height );
	
		// copy results back to h_C.
		hipMemcpy( h_Image, d_Image, 3 * sizeof(float) * width * height, hipMemcpyDeviceToHost);
		checkCUDAError("ProcessImage: memcpy2");

		for(int i=0; i<3*width*height; i++)
		{
			// d_Image, 3 kanall� oldu�u i�in do�rudan imageData'ya h_Image �zerinden kopyalan�r.
			*(imageData + i) = satchar(*(h_Image + i) * 255);
		}
	}

	virtual void ReleaseProcessing()
	{
		SingleCudaTexProcessor::ReleaseProcessing();

		hipUnbindTexture( &tex );
		checkCUDAError("unbind tex");

		hipFreeArray( cu_array );
		checkCUDAError("free device tex array");

		hipFree( d_Image );
		checkCUDAError("free device image");
	
		hipHostFree( h_Image );
		checkCUDAError("free host image");
	}

};