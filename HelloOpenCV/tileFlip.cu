#include "hip/hip_runtime.h"
#include "tileFlip.h"

/**
	\file tileFlip.cu
	CUDA tile flip kernelinin launcher metodunu ve kernelini tan�mlar.
*/

/** Kernel 1 griddeki blok boyutu ( BLOCK_SIZE x BLOCK_SIZE kare bloklar ). */
#define BLOCK_SIZE (32)

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 1

/**	
	G�r�nt�y� blok blok �eviren kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.

	*/__global__
void gpuTileFlip(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y; // sat�r No.

	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x; // s�tun No.

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	/*
	       *( image + linearIndex ): Blue, in [0, 1]
		   *( image + linearIndex + 1 ): Green, in [0, 1]
		   *( image + linearIndex + 2 ): Red, in [0, 1]
	*/

	__shared__ float smBlockB[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float smBlockG[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float smBlockR[BLOCK_SIZE][BLOCK_SIZE];

	smBlockB[threadIdx.x][threadIdx.y] = image[ cIdx ];
	smBlockG[threadIdx.x][threadIdx.y] = image[ cIdx + 1 ];
	smBlockR[threadIdx.x][threadIdx.y] = image[ cIdx + 2 ];

	__syncthreads();	
	
	image[ cIdx ]     =	smBlockB[threadIdx.y][threadIdx.x];
	image[ cIdx + 1 ] = smBlockG[threadIdx.y][threadIdx.x];
	image[ cIdx + 2 ] = smBlockR[threadIdx.y][threadIdx.x];
		
	
	//image[ cIdxRight + 2 ] = 0;

	/**( image + cIdx ) = abs((*( image + cIdx ) - *( image + cIdxRight )));
	*( image + cIdx + 1 ) = abs((*( image + cIdx + 1 ) - *( image + cIdxRight + 1 )));
	*( image + cIdx + 2 ) = abs((*( image + cIdx + 2 ) - *( image + cIdxRight + 2 )));*/
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuTileFlip kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceTileFlipLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif
	
    gpuTileFlip<<< dimGrid, dimBlock >>>( d_Image, width, height);
	
#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}
