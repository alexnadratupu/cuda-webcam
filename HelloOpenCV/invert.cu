#include "hip/hip_runtime.h"
#include "invert.h"

/**
	\file invert.cu
	CUDA invert kernelinin launcher metodunu ve kernelini tan�mlar.
*/

/** Kernel 1 griddeki blok boyutu ( BLOCK_SIZE x BLOCK_SIZE kare bloklar ). */
#define BLOCK_SIZE (32)

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 0

/**	
	G�r�nt�n�n tersini alan kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.

	*/
__global__
void gpuInvert(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	// normalize edilmi� pikselleri 1'den ��kartt���m�zda g�r�nt�n�n negatifini alm�� oluruz.
	*( image + cIdx     ) = 1 - *( image + cIdx     ); // Blue kanal�
	*( image + cIdx + 1 ) = 1 - *( image + cIdx + 1 ); // Green kanal�
	*( image + cIdx + 2 ) = 1 - *( image + cIdx + 2 ); // Red kanal�
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuInvert kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceInvertLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuInvert<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuInvert kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}
